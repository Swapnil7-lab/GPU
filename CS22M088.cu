#include "hip/hip_runtime.h"
/*
	CS 6023 Assignment 3.
	Do not make any changes to the boiler plate code or the other files in the folder.
	Use hipFree to deallocate any memory not in usage.
	Optimize as much as possible.
 */

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>

__global__
void dCreateFinalScene(int *dFinalPng, int *dOpacityMap, int *dOpacity, int* dMesh,
    int *dGlobalCoordinatesX, int *dGlobalCoordinatesY, int *dFrameSizeX,
    int *dFrameSizeY, long int* d_translations_final, int *globalFrameSizeX, int *globalFrameSizeY, int *tid){
      printf("tid: %d\n", *tid);
      printf("%d\n", *dGlobalCoordinatesX);
      printf("%d\n", *dGlobalCoordinatesY);
      printf("%ld\n", *(d_translations_final+ (*tid) * 4 + 0));
      printf("%ld\n", *(d_translations_final+ (*tid) * 4 + 1));
      printf("%ld\n", *(d_translations_final+ (*tid) * 4 + 2));
      printf("%ld\n", *(d_translations_final+ (*tid) * 4 + 3));

      (*dGlobalCoordinatesX) -= *(d_translations_final+ (*tid) * 4 + 0);//up
      (*dGlobalCoordinatesX) += *(d_translations_final+ (*tid) * 4 + 1);//down
      (*dGlobalCoordinatesY) -= *(d_translations_final+ (*tid) * 4 + 2);//left
      (*dGlobalCoordinatesY) += *(d_translations_final+ (*tid) * 4 + 3);//right


      printf("%d\n", *dFrameSizeX);
      printf("%d\n", *dFrameSizeY);
      printf("%d\n", *dGlobalCoordinatesX);
      printf("%d\n", *dGlobalCoordinatesY);
      printf("%d\n", *globalFrameSizeX);
      printf("%d\n", *globalFrameSizeY);


      for(int i = 0; i < (*dFrameSizeX); i++){
        for(int j = 0; j < (*dFrameSizeY); j++){
          if((*dGlobalCoordinatesX) + i >= 0 && (*dGlobalCoordinatesX) + i < (*globalFrameSizeX)){
            if((*dGlobalCoordinatesY) + j >= 0 && (*dGlobalCoordinatesY) + j < (*globalFrameSizeY)){
              if(*(dOpacityMap + ((*dGlobalCoordinatesX) + i) * (*globalFrameSizeY) + (*dGlobalCoordinatesY) + j) < (*dOpacity)){
                *(dOpacityMap + ((*dGlobalCoordinatesX) + i) * (*globalFrameSizeY) + (*dGlobalCoordinatesY) + j) = (*dOpacity);
                *(dFinalPng + ((*dGlobalCoordinatesX) + i) * (*globalFrameSizeY) +  (*dGlobalCoordinatesY) + j) =
                *(dMesh + (i * (*dFrameSizeY)) + j);
              }
            }

          }
        }
      }
    }

__global__
void dcalcTranslations(int* dParent, long int* dTranslations_final, int *V, int *done){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;


  if(tid < *V){
    printf("tid: %d\n", tid);

    while(done[tid] == 0){
      if(done[dParent[tid]] == 1){
        printf("parent of %d id %d", tid, dParent[tid]);
        *(dTranslations_final + (tid * 4) + 0) += *(dTranslations_final + (dParent[tid] * 4) + 0);
        *(dTranslations_final + (tid * 4) + 1) += *(dTranslations_final + (dParent[tid] * 4) + 1);
        *(dTranslations_final + (tid * 4) + 2) += *(dTranslations_final + (dParent[tid] * 4) + 2);
        *(dTranslations_final + (tid * 4) + 3) += *(dTranslations_final + (dParent[tid] * 4) + 3);
        
        done[tid] = 1;
      }
    }
  }
}


void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input.
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;


	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ;
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ;
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}


void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL;
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}

	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}


int main (int argc, char **argv) {

	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ;

	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;

	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now () ;


	// Code begins here.
	// Do not change anything above this comment.
  int h_parent[V];
  int h_done[V];
  h_parent[0] = 0;

  for(int i = 0; i < edges.size(); i++){
    h_parent[edges[i][1]] = edges[i][0];
  }
  for(int i = 0; i < V; i++){
    h_done[i] = 0;
  }
  h_done[0] = 1;

  long int h_translations_final[V*4] = {0};

  for(int i = 0; i < translations.size(); i++){//1 2 4
    std::vector<int> translation = translations[i];
    h_translations_final[translation[0] * 4 + translation[1]] += translation[2];
  }

  int *dParent;
  long int *dTranslations_final;
  int *total_nodes;
  int* d_done;
  int *dFinalPng;
  int *dOpacityMap;

  hipMalloc(&dParent, (V) * sizeof(int));
  hipMalloc(&dTranslations_final, (V * 4) * sizeof(long int));
  hipMalloc(&total_nodes, sizeof(int));
  hipMalloc(&d_done, V * sizeof(int));
  hipMalloc(&dFinalPng, sizeof (int) * frameSizeX * frameSizeY);
  hipMalloc(&dOpacityMap, sizeof (int) * frameSizeX * frameSizeY);

  hipMemcpy(dParent, h_parent, V * sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dTranslations_final, h_translations_final, (4 * V) * sizeof(long int),hipMemcpyHostToDevice);
  hipMemcpy(total_nodes, &V, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_done,h_done,V * sizeof(int),hipMemcpyHostToDevice);
  hipMemset(dOpacityMap, INT_MIN, (frameSizeX * frameSizeY) * sizeof(int));
  hipMemset(dFinalPng, 0, (frameSizeX * frameSizeY) * sizeof(int));
  //transitions
  dcalcTranslations<<<ceil(V/1024.0), 1024>>>(dParent, dTranslations_final, total_nodes, d_done);
  hipDeviceSynchronize();
  hipMemcpy(h_translations_final, dTranslations_final, (4 * V) * sizeof(long int), hipMemcpyDeviceToHost);

  for(int i = 0; i < V; i++)
  printf("%ld %ld %ld %ld\n", *(h_translations_final + (i * 4) + 0), *(h_translations_final + (i * 4) + 1), *(h_translations_final + (i * 4) + 2), *(h_translations_final + (i * 4) + 3));

  //scene
  for(int i = 0; i < V; i++){
    int *dOpacity;
    int *dMesh;
    int *dGlobalCoordinatesX;
    int *dGlobalCoordinatesY;
    int *dFrameSizeX;
    int *dFrameSizeY;
    int *globalFrameSizeX;
    int *globalFrameSizeY;
    int *tid;

    hipMalloc(&tid, sizeof(int));
    hipMalloc(&globalFrameSizeX, sizeof(int));
    hipMalloc(&globalFrameSizeY, sizeof(int));
    hipMalloc(&dOpacity, sizeof(int));
    hipMalloc(&dGlobalCoordinatesX, sizeof(int));
    hipMalloc(&dGlobalCoordinatesY, sizeof(int));
    hipMalloc(&dFrameSizeX, sizeof(int));
    hipMalloc(&dFrameSizeY, sizeof(int));
    hipMalloc(&dMesh, (hFrameSizeX[i] * hFrameSizeY[i]) * sizeof(int));

    hipMemcpy(dOpacity, &hOpacity[i], sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(dGlobalCoordinatesX, &hGlobalCoordinatesX[i], sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(dGlobalCoordinatesY, &hGlobalCoordinatesY[i], sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(dFrameSizeX, &hFrameSizeX[i], sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(dFrameSizeY, &hFrameSizeY[i], sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(dMesh, hMesh[i], (hFrameSizeX[i] * hFrameSizeY[i]) * sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(tid, &i, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(globalFrameSizeX, &frameSizeX, sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(globalFrameSizeY, &frameSizeY, sizeof(int), hipMemcpyHostToDevice);


    dCreateFinalScene<<<1,1>>>(dFinalPng, dOpacityMap, dOpacity, dMesh,
    dGlobalCoordinatesX, dGlobalCoordinatesY, dFrameSizeX,
    dFrameSizeY, dTranslations_final, globalFrameSizeX, globalFrameSizeY, tid);
    hipDeviceSynchronize();
  }

  hipMemcpy(hFinalPng, dFinalPng, (frameSizeX * frameSizeY) * sizeof(int),hipMemcpyDeviceToHost);

	// Do not change anything below this comment.
	// Code ends here.

	auto end  = std::chrono::high_resolution_clock::now () ;

	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken.count()) ;
	// Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;

}
