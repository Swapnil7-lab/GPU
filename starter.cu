/**
*   CS6023: GPU Programming 
*   Assignment 1
*   
*   Please don't change any existing code in this file.
*
*   You can add your code whereever needed. Please add necessary memory APIs
*   for your implementation. Use cudaFree() to free up memory as soon as you're
*   done with an allocation. This will ensure that you don't run out of memory 
*   while running large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;


__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int row = id / N;
	int col = id % N;

	if(row < N && col < N){
		A[row * N + col] = A[row * N + col] * B[col * N + row];
	}

}

__global__
void FindWeightMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
	int id = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.y + threadIdx.y;
	int row = id / N;
	int col = id % N;

	if(row < N && col < N){
		A[row * N + col] = max(A[row * N + col], B[row * N + col]);
	}
}

__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
	int blockNum = blockIdx.x * gridDim.y + blockIdx.y;
    int threadNum = blockNum * (blockDim.x * blockDim.y) + threadIdx.x * blockDim.y + threadIdx.y;

    int row = threadNum / (2 * N);
    int col = threadNum % (2 * N);

    if(row < 2 * N && col < 2 * N){
      int rowA = row % N;
      int colA = col % N;
      B[row * 2 * N + col] = B[row * 2 * N + col] * A[rowA * N + colA];
    }
}


int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    */

    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;
	
	hipMalloc(&d_A, N * N * sizeof(long int));
	hipMalloc(&d_B, N * N * sizeof(long int));
	hipMalloc(&d_C, N * N * sizeof(long int));
	hipMalloc(&d_D, 2 * N * 2 * N * sizeof(long int));
	
	hipMemcpy(d_A, A, N * N * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N * N * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, N * N * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_D, D, 2 * N * 2 * N * sizeof(long int), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);


    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;

	hipFree(d_B);

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);


    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;

	hipFree(d_C);
	
    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);


    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;
	
	hipFree(d_A);

    // Make sure your final output from the device is stored in d_D.

    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}